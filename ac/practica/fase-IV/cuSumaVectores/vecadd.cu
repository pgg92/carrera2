#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.
 *
 * This software and the information contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a Non-Disclosure Agreement.  Any reproduction or
 * disclosure to any third party without the express written consent of
 * NVIDIA is prohibited.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.  This source code is a "commercial item" as
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer software" and "commercial computer software
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

#define VECTOR_ELEMENTS 30000000
#define COMPUTE_N_ELEMENTS_PER_THREAD 1

// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "vecadd.h"
#include <Windows.h>

#define DEBUG_ 1

typedef LARGE_INTEGER timeStamp;
double getTime();

__global__ void vecadd(float* C, const float* A, const float* B)
{
    // ===================================================================
    // Calcula el �ndice para acceder a cada elemento
    // Calcula la suma de las posiciones correspondientes y almacena el resultado
    // ===================================================================

    const int i = (threadIdx.x + blockIdx.x * blockDim.x) * COMPUTE_N_ELEMENTS_PER_THREAD;
	if( i < VECTOR_ELEMENTS )
	{
		for(int j=0; j < COMPUTE_N_ELEMENTS_PER_THREAD; j++)
		{
			C[i+j] = A[i+j] + B[i+j];
		}
	}
}

#define ERROR_CHECK { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
    printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__);}}

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
    runTest(argc, argv);
		getchar();
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{
    float *A, *B, *C, *reference;
    float *A_d, *B_d, *C_d;

	double gpu_start_time_trans, gpu_end_time_trans;
	double gpu_start_time, gpu_end_time;
	double cpu_start_time, cpu_end_time;

    // Allocate input vectors (host memory)
    A = (float *) malloc(VECTOR_ELEMENTS * sizeof(float));
    assert(A != NULL);
    B = (float *) malloc(VECTOR_ELEMENTS * sizeof(float));
    assert(B != NULL);

    printf("  Allocate host memory for input vectors A, B\n");
    printf("    A: %d elements\n", VECTOR_ELEMENTS);
    printf("    B: %d elements\n", VECTOR_ELEMENTS);

	printf(" Required memory for %d elements: %d MB ",VECTOR_ELEMENTS, (VECTOR_ELEMENTS*3*sizeof(float)) / (1024*1024) );

    // Allocate output vector (host memory)
    C = (float *) malloc(VECTOR_ELEMENTS * sizeof(float));
    assert(C != NULL);

	#ifdef DEBUG_
		printf("  Allocate memory for the output vector on host side.\n");
	#endif

	    // Initialize input vectors with some values
    for (int i = 0; i < VECTOR_ELEMENTS; i++) {
        A[i] = 1.f;
        B[i] = 2.f;
    }

    // Allocate input vectors (device memory)
    hipMalloc((void **) &A_d, VECTOR_ELEMENTS * sizeof(float));
    hipMalloc((void **) &B_d, VECTOR_ELEMENTS * sizeof(float));

	#ifdef DEBUG_
		printf("  Allocate device memory.\n");
	#endif

	gpu_start_time_trans = getTime();

    // Allocate output vector (device memory)
    hipMalloc((void **) &C_d, VECTOR_ELEMENTS * sizeof(float));

	#ifdef DEBUG_
		printf("  Allocate device memory for output vector.\n");
	#endif

    // Copy input vectors to device memory
    hipMemcpy(A_d, A, VECTOR_ELEMENTS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, VECTOR_ELEMENTS * sizeof(float), hipMemcpyHostToDevice);

    #ifdef DEBUG_
		printf("  Copy host memory to device.\n");
		printf("  Setup kernel execution parameters.\n");
	#endif

    // ===================================================================
    // Calcula las dimensiones del grid e invoca el kernel
    // ===================================================================
    dim3 block(512);
    dim3 grid( (VECTOR_ELEMENTS + (block.x*COMPUTE_N_ELEMENTS_PER_THREAD-1) )/ (block.x*COMPUTE_N_ELEMENTS_PER_THREAD) );

	#ifdef DEBUG_
		printf("  # of threads in a block: %d\n", block.x);
		printf("  # of blocks in a grid  : %d\n", grid.x);
		printf("  Executing the kernel...\n");
	#endif

	gpu_start_time = getTime();

    vecadd<<<grid,block>>>(C_d,A_d,B_d);

    hipDeviceSynchronize();
	gpu_end_time = getTime();
    ERROR_CHECK

    hipMemcpy(C, C_d, VECTOR_ELEMENTS * sizeof(float), hipMemcpyDeviceToHost);

	gpu_end_time_trans = getTime();

	#ifdef DEBUG_
		printf("  Copy result from device to host.\n");
	#endif

    // Compare results
    {
        printf("\nCheck results with those computed by CPU.\n");
        printf ("  Computing reference solution.\n");

		reference = (float *) malloc(VECTOR_ELEMENTS * sizeof(float));
        
		cpu_start_time = getTime();
		// Compute reference vector
        computeGold(reference, A, B, VECTOR_ELEMENTS);
		cpu_end_time = getTime();

        // Compare results
        bool correct = true;
		for( int j = 0; j < 10; j++)
		{
        for (int i = 0; i < VECTOR_ELEMENTS; i++) {
            if (reference[i] != C[i]) {
                correct = false;
                break;
            }
        }
		}

        if (correct == true) {
            printf("  test: passed!\n");
        } else {
            printf("  test: failed!\n");
        }
		free(reference);
    }

	printf("Tiempo ejecuci�n GPU (sin incluir transferencia de datos): %fs\n", \
	 gpu_end_time - gpu_start_time);
	printf("Tiempo ejecuci�n GPU (con transferencia de datos): %fs\n", \
	 gpu_end_time_trans - gpu_start_time_trans);
	printf("Tiempo de ejecuci�n en la CPU                          : %fs\n", \
	 cpu_end_time - cpu_start_time);


	printf("Se ha conseguido un factor de aceleraci�n %fx utilizando CUDA (sin considerar transferencias) \n", (cpu_end_time - cpu_start_time) / (gpu_end_time - gpu_start_time) );
    // clean up memory
    free(A);
    free(B); 
    free(C);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

/* Funciones auxiliares */
double getTime()
{
	timeStamp start;
	timeStamp dwFreq;
	QueryPerformanceFrequency(&dwFreq);
	QueryPerformanceCounter(&start);
	return double(start.QuadPart) / double(dwFreq.QuadPart);
}

