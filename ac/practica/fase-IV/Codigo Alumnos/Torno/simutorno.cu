#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------------*/
/*  FICHERO:       simutorno.cu									          */
/*  AUTOR:         Antonio Jimeno											  */
/*													                          */
/*  RESUMEN												                      */
/*  ~~~~~~~												                      */
/* Ejercicio grupal para simulaci�n del movimiento de una herramienta         */
/* tipo torno utilizando GPUs                                                 */
/*----------------------------------------------------------------------------*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>


// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "simutorno.h"
#include <Windows.h>



#define ERROR_CHECK { hipError_t err; if ((err = hipGetLastError()) != hipSuccess) { printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__);}}

typedef LARGE_INTEGER timeStamp;
double getTime();

/*----------------------------------------------------------------------------*/
/*  FUNCION A PARALELIZAR  (versi�n secuencial-CPU)  				          */
/*	Simula el movimiento de una herramienta de una m�quina tipo torno         */
/*  Realiza pasossim pasos de simulacion sobre una superficie definida por    */
/*  una malla de vtotal x utotal puntos 3D                                    */
/*  El helicoide se define con un paso de PasoHelicoide mil�metros y          */
/*  con una definici�n de PuntosVueltaHelicoide por cada giro completo        */
/*----------------------------------------------------------------------------*/
int SimulacionTornoCPU(int pasossim)
{
	/* Parametros de mecanizado */
	double posX = 0.0;								// Posici�n de la herramienta
	double angle = 0.0;								// Posici�n del eje de giro
	double incX = PasoHelicoide / (double)PuntosVueltaHelicoide;
													// Incremento de la posici�n de la herramienta en cada paso de simulaci�n
	double incA = 360.0 / (double)PuntosVueltaHelicoide;
													// Incremento angular del eje de giro a cada paso de simulaci�n

	for (int i = 0; i < pasossim; i++)				// Bucle de simulaci�n
	{
		double AvanceMin = 1e10;					// Inicializaci�n del m�nimo
		for (int u = 0; u<S.UPoints; u++)			// Recorrido de todos los puntos de la superficie para cada paso de simulaci�n
		{
			for (int v = 0; v<S.VPoints; v++)
			{
				double px = S.Buffer[v][u].x + posX;// Cambio de la posici�n de la superficie por traslaci�n de la herramienta
													// Cambio de la posici�n de la superficie por giro sobre el eje X 
				double py = S.Buffer[v][u].y * cos(angle*M_PI_180) - S.Buffer[v][u].z * sin(angle*M_PI_180);
				double pz = S.Buffer[v][u].y * sin(angle*M_PI_180) + S.Buffer[v][u].z * cos(angle*M_PI_180);
													// Calcula la distancia del punto transformado a la herramienta situada en (0,TOOLYPOS,0)
				double modulo = sqrt(px*px + (py - TOOLYPOS)*(py - TOOLYPOS) + pz*pz);
													// Si el punto est� en la zona de influencia de la herramienta (-TOOLWIDTH<px<+TOOLWIDTH) y es el m�s cercano se guardan sus coordenadas U y V
				if ((fabs(px))<TOOLWIDTH && (modulo<AvanceMin))
				{
					AvanceMin = modulo; GanadorUCPU[i] = u; GanadorVCPU[i] = v;
				}
			}
		}
													// Se actualiza la posici�n para el nuevo paso de simulaci�n
		angle += incA;
		posX -= incX;
	}
	return OKSIM;									// Simulaci�n CORRECTA
}

// ---------------------------------------------------------------
// ---------------------------------------------------------------
// FUNCION A IMPLEMENTAR POR EL GRUPO (paralelizaci�n de SimulacionTornoCPU)
// ---------------------------------------------------------------
// ---------------------------------------------------------------

 int SimulacionTornoGPU(int pasossim)
{
	 return OKSIM;
}
 // ---------------------------------------------------------------
 // ---------------------------------------------------------------
 // ---------------------------------------------------------------
 // ---------------------------------------------------------------
 // ---------------------------------------------------------------

 // Declaraciones adelantadas de funciones
 int LeerSuperficie(const char *fichero);



////////////////////////////////////////////////////////////////////////////////
//PROGRAMA PRINCIPAL
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{


	double gpu_start_time, gpu_end_time;
	double cpu_start_time, cpu_end_time;

	/* Numero de argumentos */
	if (argc != 3)
	{
		fprintf(stderr, "Numero de parametros incorecto\n");
		fprintf(stderr, "Uso: %s superficie pasossim\n", argv[0]);
		return;
	}

	/* Apertura de Fichero */
	printf("Prueba simulaci�n torno...\n");
	/* Datos de la superficie */
	if (LeerSuperficie((char *)argv[1]) == ERRORSIM)
	{
		fprintf(stderr, "Lectura de superficie incorrecta\n");
		return;
	}
	int pasossim = atoi((char *)argv[2]);
	// Creaci�n buffer resultados para versiones CPU y GPU
	GanadorVCPU = (int*)malloc(pasossim*sizeof(int));
	GanadorUCPU = (int*)malloc(pasossim*sizeof(int));
	GanadorVGPU = (int*)malloc(pasossim*sizeof(int));
	GanadorUGPU = (int*)malloc(pasossim*sizeof(int));

	/* Algoritmo a paralelizar */
	cpu_start_time = getTime();
	if (SimulacionTornoCPU(pasossim) == ERRORSIM)
	{
		fprintf(stderr, "Simulaci�n CPU incorrecta\n");
		BorrarSuperficie();
		if (GanadorVCPU != NULL) free(GanadorVCPU);
		if (GanadorUCPU != NULL) free(GanadorUCPU);
		if (GanadorVGPU != NULL) free(GanadorVGPU);
		if (GanadorUGPU != NULL) free(GanadorUGPU);		exit(1);
	}
	cpu_end_time = getTime();
	/* Algoritmo a implementar */
	gpu_start_time = getTime();
	if (SimulacionTornoGPU(pasossim) == ERRORSIM)
	{
		fprintf(stderr, "Simulaci�n GPU incorrecta\n");
		BorrarSuperficie();
		if (GanadorVCPU != NULL) free(GanadorVCPU);
		if (GanadorUCPU != NULL) free(GanadorUCPU);
		if (GanadorVGPU != NULL) free(GanadorVGPU);
		if (GanadorUGPU != NULL) free(GanadorUGPU);
		return;
	}
	gpu_end_time = getTime();
	// Comparaci�n de correcci�n
	int comprobar = OKSIM;
	for (int i = 0; i<pasossim; i++)
	{
		if ((GanadorVCPU[i] != GanadorVGPU[i]) || (GanadorUCPU[i] != GanadorUGPU[i]))
		{
			comprobar = ERRORSIM;
			fprintf(stderr, "Fallo en paso %d de simulaci�n, valor correcto V=%d U=%d\n", i, GanadorVCPU[i], GanadorUCPU[i]);
		}
	}
	// Impresion de resultados
	if (comprobar == OKSIM)
	{
		printf("Simulaci�n correcta!\n");

	}
	// Impresi�n de resultados
	printf("Tiempo ejecuci�n GPU : %fs\n", \
		gpu_end_time - gpu_start_time);
	printf("Tiempo de ejecuci�n en la CPU : %fs\n", \
		cpu_end_time - cpu_start_time);
	printf("Se ha conseguido un factor de aceleraci�n %fx utilizando CUDA\n", (cpu_end_time - cpu_start_time) / (gpu_end_time - gpu_start_time));
	// Limpieza de buffers
	BorrarSuperficie();
	if (GanadorVCPU != NULL) free(GanadorVCPU);
	if (GanadorUCPU != NULL) free(GanadorUCPU);
	if (GanadorVGPU != NULL) free(GanadorVGPU);
	if (GanadorUGPU != NULL) free(GanadorUGPU);
	return;
}

int
main(int argc, char** argv)
{
	runTest(argc, argv);
	getchar();
}

/* Funciones auxiliares */
double getTime()
{
	timeStamp start;
	timeStamp dwFreq;
	QueryPerformanceFrequency(&dwFreq);
	QueryPerformanceCounter(&start);
	return double(start.QuadPart) / double(dwFreq.QuadPart);
}



/*----------------------------------------------------------------------------*/
/*	Funci�n:  LeerSuperficie(char *fichero)						              */
/*													                          */
/*	          Lee los datos de la superficie de un fichero con formato .FOR   */
/*----------------------------------------------------------------------------*/
int LeerSuperficie(const char *fichero)
{
	int i, j, count;		/* Variables de bucle */
	int utotal,vtotal;		/* Variables de tama�o de superficie */
	FILE *fpin; 			/* Fichero */
	char cadena[255];
	double x, y, z;

	cadena[0] = 0;
	/* Apertura de Fichero */
	if ((fpin = fopen(fichero, "r")) == NULL) return ERRORSIM;
	/* Lectura de cabecera */
	while (!feof(fpin) && strcmp(cadena, "[HEADER]")) fscanf(fpin, "%s\n", cadena);
	if (fscanf(fpin, "SECTION NUMBER=%d\n", &utotal)<0) return ERRORSIM;
	if (fscanf(fpin, "POINTS PER SECTION=%d\n", &vtotal)<0) return ERRORSIM;
	if (fscanf(fpin, "STEP=%lf\n", &PasoHelicoide)<0) return ERRORSIM;
	if (fscanf(fpin, "POINTS PER ROUND=%d\n", &PuntosVueltaHelicoide)<0) return ERRORSIM;
	if (utotal*vtotal <= 0) return ERRORSIM;
	/* Localizacion de comienzo */
	while (!feof(fpin) && strcmp(cadena, "[GEOMETRY]")) fscanf(fpin, "%s\n", cadena);
	if (feof(fpin)) return ERRORSIM;
	/* Inicializaci�n de parametros geometricos */
	if (CrearSuperficie(utotal, vtotal) == ERRORSIM) return ERRORSIM;
	/* Lectura de coordenadas */
	count = 0;
	for (i = 0; i<utotal; i++)
	{
		for (j = 0; j<vtotal; j++)
		{
			if (!feof(fpin))
			{
				fscanf(fpin, "%lf %lf %lf\n", &x, &y, &z);
				S.Buffer[j][i].x = x;
				S.Buffer[j][i].y = y;
				S.Buffer[j][i].z = z;
				count++;
			}
			else break;
		}
	}
	fclose(fpin);
	if (count != utotal*vtotal) return ERRORSIM;
	return OKSIM;
}



