#include "hip/hip_runtime.h"
/*
* Ejemplo en CUDA de la implementaci�n background subtraction
* aplicado a dos im�genes
*
* Adaptacion de un codigo de calculo de mediana
* desarrollado por Sergio Orts-Escolano
* Copyright Universidad de Alicante, 2012
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#if _WIN32
#include <Windows.h>
#else
#include <sys/types.h>
#include <sys/time.h>
#endif

// Ficheros de inclusi�n para que funcione el intellisense en Visual Studio
#include "hip/hip_runtime.h"
#include ""

#include "EasyBMP.h"

#if _WIN32
typedef LARGE_INTEGER timeStamp;
void getCurrentTimeStamp(timeStamp& _time);
timeStamp getCurrentTimeStamp();
double getTimeMili(const timeStamp& start, const timeStamp& end);
double getTimeSecs(const timeStamp& start, const timeStamp& end);
#endif

// Dimensiones de la imagen a procesar
int WIDTH;
int HEIGHT;

// valor del umbral 
int Threshold;


// Funciones auxiliares
double get_current_time();
void checkCUDAError(const char*);


// Tama�o de grid y bloque CUDA
#define GRID_W  32
#define GRID_H  32
#define BLOCK_W 32
#define BLOCK_H 32

// Buffers de imagenes
unsigned char *input_imageb;
unsigned char *input_imagef;
unsigned char *gpu_output;




// CUDA kernel background
__global__ void Background(unsigned char *d_output, unsigned char *d_inputb, unsigned char *d_inputf, int width)
{

	/*
	* Calculamos la fila y columna global para este hilo
	*/
	// TO DO


	// Realizar la substracci�n para el pixel correspondiente a este hilo 
	// TO DO


}

/***********************************************************************************/

// El main puede tener como argumentos: nombres de los fichero de las imagenes (tiene que ser BMP) y el umbral
int main(int argc, char *argv[])
{
	int x, y, xWindow, yWindow, x2, y2;
	int i;
	int errors;
	int temp, mean, clr;

	double start_time_inc_data, end_time_inc_data;
	double cpu_start_time, cpu_end_time;

	unsigned char *d_inputb, *d_inputf, *d_output;

	unsigned char *output_image;
	unsigned char *imagen_resultado;
	int rows;
	int cols;

	// Leemos las im�genes 
	BMP Fondo, Image;
	Fondo.ReadFromFile("background.bmp");
	Image.ReadFromFile("imagen.bmp");
	//Fondo.ReadFromFile(argv[1]);
	//Image.ReadFromFile(argv[2]);

	BMP Resultado;

	// Leemos el valor del umbral
	Threshold = 120;
	//Threshold = atoi(argv[3]);

	// Calculo del tama�o de la imagen
	WIDTH = Fondo.TellWidth();
	HEIGHT = Fondo.TellHeight();

	// Establecemos el tama�o de la imagen de salida
	Resultado.SetSize(Fondo.TellWidth(), Fondo.TellHeight());
	Resultado.SetBitDepth(1);

	// Reserva memoria en el host para alojar la imagen
	input_imageb = (unsigned char*)calloc(((HEIGHT * WIDTH) * 1), sizeof(unsigned char));
	input_imagef = (unsigned char*)calloc(((HEIGHT * WIDTH) * 1), sizeof(unsigned char));
	gpu_output = (unsigned char*)calloc(((HEIGHT * WIDTH) * 1), sizeof(unsigned char));
	output_image = (unsigned char*)calloc(((WIDTH * HEIGHT) * 1), sizeof(unsigned char));


	for (int i = 0; i < WIDTH; i++)
	{
		for (int j = 0; j < HEIGHT; j++)
		{
			input_imageb[i*WIDTH + j] = Fondo(i, j)->Red;
			input_imagef[i*WIDTH + j] = Image(i, j)->Red;
		}
	}


	/*
	* Versi�n CPU background
	*/

	printf("Grid size: %dx%d\n", GRID_W, GRID_H);
	printf("Block size: %dx%d\n", BLOCK_W, BLOCK_H);

	// Calculamos memoria necesaria para alojar las imagenes 
	size_t memSize = WIDTH * HEIGHT * sizeof(unsigned char);

	/* Reservamos memoria en la GPU */
	hipMalloc(&d_inputb, memSize);
	hipMalloc(&d_inputf, memSize);
	hipMalloc(&d_output, memSize);


	start_time_inc_data = get_current_time();

	/*
	* Copiamos todos los arrays a la memoria de la GPU
	*/
	hipMemcpy(d_inputb, input_imageb, memSize, hipMemcpyHostToDevice);
	hipMemcpy(d_inputf, input_imagef, memSize, hipMemcpyHostToDevice);
	hipMemcpy(d_output, input_imagef, memSize, hipMemcpyHostToDevice);

	/***********************************************************/
	// Ejecutar background en la GPU
	/* Ejecuci�n kernel  */
	// TO DO - Calcular tama�o de bloque y grid para la correcta ejecucion del kernel
	// TO DO - Ejecutar el kernel


	// Copiamos de la memoria de la GPU 
	hipMemcpy(gpu_output, d_output, memSize, hipMemcpyDeviceToHost);

	end_time_inc_data = get_current_time();

	checkCUDAError("Background CUDA: ");

	/*************************************************************/

	/****************************
	* Versi�n CPU background
	*****************************/
	cpu_start_time = get_current_time();


	for (y = 1; y < HEIGHT - 1; y++) {
		for (x = 1; x < WIDTH - 1; x++) {

			clr = 1;
			mean = 0;
			for (yWindow = -1; yWindow < 2; yWindow++) {
				y2 = y + yWindow;
				for (xWindow = -1; xWindow <2; xWindow++) {
					x2 = x + xWindow;
					mean += input_imagef[y2*WIDTH + x2];
				}
			}

			mean = mean / 9;
			temp = abs((mean - input_imageb[y*WIDTH + x]));

			if (temp > Threshold)
				clr = 0;
			if (clr == 0)
				output_image[y*WIDTH + x] = 255;
			else
				output_image[y*WIDTH + x] = 0;


		}
	}


	cpu_end_time = get_current_time();

	/* Comprobamos que los resultados de la GPU coinciden con los calculados en la CPU */

	errors = 0;
	for (y = 1; y < HEIGHT - 1; y++) {
		for (x = 1; x < WIDTH - 1; x++) {
			if (output_image[y *WIDTH + x] != gpu_output[y *WIDTH + x]) {
				errors++;
				printf("Error en %d,%d (CPU=%i, GPU=%i)\n", x, y, \
					output_image[y *WIDTH + x], \
					gpu_output[y*WIDTH + x]);
			}
		}
	}

	if (errors == 0) printf("\n\n ***TEST CORRECTO*** \n\n\n");

	imagen_resultado = (unsigned char*)calloc(((WIDTH * HEIGHT) * 1), sizeof(unsigned char));


	for (y = 0; y < HEIGHT; y++) {
		for (x = 0; x < WIDTH; x++) {
			imagen_resultado[y*WIDTH + x] = gpu_output[y*WIDTH + x];
		}
	}

	hipFree(d_inputb);
	hipFree(d_inputf);
	hipFree(d_output);

	printf("Tiempo ejecuci�n GPU (Incluyendo transferencia de datos): %fs\n", \
		end_time_inc_data - start_time_inc_data);
	printf("Tiempo de ejecuci�n en la CPU                          : %fs\n", \
		cpu_end_time - cpu_start_time);

	// Copiamos el resultado al formato de la libreria y guardamos el fichero BMP procesado
	for (int i = 0; i < WIDTH; i++)
	{
		for (int j = 0; j < HEIGHT; j++)
		{
			Resultado(i, j)->Red = imagen_resultado[i*WIDTH + j];
			Resultado(i, j)->Green = imagen_resultado[i*WIDTH + j];
			Resultado(i, j)->Blue = imagen_resultado[i*WIDTH + j];
		}
	}
	// Guardamos el resultado de aplicar el filtro en un nuevo fichero
	Resultado.WriteToFile("resultado_background.bmp");

	getchar();
	return 0;
}


/* Funciones auxiliares */

#if _WIN32
void getCurrentTimeStamp(timeStamp& _time)
{
	QueryPerformanceCounter(&_time);
}

timeStamp getCurrentTimeStamp()
{
	timeStamp tmp;
	QueryPerformanceCounter(&tmp);
	return tmp;
}

double getTimeMili()
{
	timeStamp start;
	timeStamp dwFreq;
	QueryPerformanceFrequency(&dwFreq);
	QueryPerformanceCounter(&start);
	return double(start.QuadPart) / double(dwFreq.QuadPart);
}
#endif 

double get_current_time()
{
#if _WIN32 
	return getTimeMili();
#else
	static int start = 0, startu = 0;
	struct timeval tval;
	double result;

	if (gettimeofday(&tval, NULL) == -1)
		result = -1.0;
	else if (!start) {
		start = tval.tv_sec;
		startu = tval.tv_usec;
		result = 0.0;
	}
	else
		result = (double)(tval.tv_sec - start) + 1.0e-6*(tval.tv_usec - startu);
	return result;
#endif
}

/* Funci�n para comprobar errores CUDA */
void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

