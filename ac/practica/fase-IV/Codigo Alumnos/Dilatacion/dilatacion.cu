#include "hip/hip_runtime.h"
/*
 * Ejemplo en CUDA de implementaci�n de una dilatacion 
 * para procesamiento de im�genes
 *
 * Adaptacion por Francisco Florez-Revuelta (2016) de un codigo de calculo de mediana
 * desarrollado por Sergio Orts-Escolano
 * Copyright Universidad de Alicante, 2012 
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#if _WIN32
	#include <Windows.h>
#else
	#include <sys/types.h>
	#include <sys/time.h>
#endif

// Ficheros de inclusi�n para que funcione el intellisense en Visual Studio
#include "hip/hip_runtime.h"
#include ""

#include "EasyBMP.h"

#if _WIN32
	typedef LARGE_INTEGER timeStamp;
	void getCurrentTimeStamp(timeStamp& _time);
	timeStamp getCurrentTimeStamp();
	double getTimeMili(const timeStamp& start, const timeStamp& end);
	double getTimeSecs(const timeStamp& start, const timeStamp& end);
#endif

// Dimensiones de la imagen a procesar
int WIDTH;
int HEIGHT;

// Funciones auxiliares
double get_current_time();
void checkCUDAError(const char*);


// Tama�o del elemento estructurante es 2*SIZE_STRUCTURING_ELEMENT+1
int SIZE_STRUCTURING_ELEMENT;

// Tama�o de grid y bloque CUDA
#define GRID_W  32
#define GRID_H  32
#define BLOCK_W 32
#define BLOCK_H 32

// Buffers con el halo correspondiente. El halo crea un borde alrededor de la imagen de color 0 que permite evitar errores por salirse de la imagen 
unsigned char *host_input;
unsigned char *gpu_output;
unsigned char *host_output;

// CUDA kernel dilatacion
__global__ void dilatacion(unsigned char *d_output, unsigned char *d_input, int width, int SIZE_STRUCTURING_ELEMENT)
{
	
	/*
	* Calculamos la fila y columna global para este hilo a partir de
	* Recuerda sumar SIZE_STRUCTURING_ELEMENT para tener en cuenta el halo introducido
	*/
	// TO DO
	

	// Obtener dilatacion para el pixel correspondiente a este hilo 
	// TO DO
	

}


// El main tiene dos argumentos: nombre del fichero de la imagen (tiene que ser BMP) y el tama�o del elemento estructurante
int main(int argc, char *argv[])
{
	int x, y, xWindow, yWindow, x2, y2;
	int i;
	int errors;
	int maxColor;

	double start_time_inc_data, end_time_inc_data;
	double cpu_start_time, cpu_end_time;

	unsigned char *d_input, *d_output, *d_edge, *tmp;

	unsigned char *input_image;
	unsigned char *output_image;
	int rows;
	int cols;

	// Leemos la imagen 
	BMP Image;
	Image.ReadFromFile(argv[1]);
	
	// Leemos el tama�o del elemento estructurante
	SIZE_STRUCTURING_ELEMENT = atoi(argv[2]);

	// Calculo del tama�o de la imagen
	WIDTH = Image.TellWidth();
	HEIGHT = Image.TellHeight();
	
	// Reserva de memoria para 
	host_input = (unsigned char*)calloc(((HEIGHT + 2 * SIZE_STRUCTURING_ELEMENT) * (WIDTH + 2 * SIZE_STRUCTURING_ELEMENT)), sizeof(unsigned char)); 
	gpu_output = (unsigned char*)calloc(((HEIGHT + 2 * SIZE_STRUCTURING_ELEMENT) * (WIDTH + 2 * SIZE_STRUCTURING_ELEMENT)), sizeof(unsigned char));
	host_output = (unsigned char*)calloc(((HEIGHT + 2 * SIZE_STRUCTURING_ELEMENT) * (WIDTH + 2 * SIZE_STRUCTURING_ELEMENT)), sizeof(unsigned char));

	// Alojamos memoria en el host para alojar la imagen
	input_image = (unsigned char*)calloc(((HEIGHT * WIDTH) * 1), sizeof(unsigned char));

	for (int i = 0; i < WIDTH; i++)
	{
		for (int j = 0; j < HEIGHT; j++)
		{
			input_image[i*WIDTH + j] = Image(i, j)->Red;
		}
	}


	/*
	* Versi�n CPU dilatacion
	*/
	
	printf("Grid size: %dx%d\n", GRID_W, GRID_H);
	printf("Block size: %dx%d\n", BLOCK_W, BLOCK_H);

	// Calculamos memoria necesaria para alojar la imagen junto con el halo
	size_t memSize = (WIDTH + 2 * SIZE_STRUCTURING_ELEMENT) * (HEIGHT + 2 * SIZE_STRUCTURING_ELEMENT) * sizeof(unsigned char);
		
	/* Reservamos memoria en la GPU */
	hipMalloc(&d_input, memSize);
	hipMalloc(&d_output, memSize);

	/* Inicializamos a cero para asegurar que el halo tiene valores correctos */
	for (y = 0; y < HEIGHT + 2*SIZE_STRUCTURING_ELEMENT; y++) {
		for (x = 0; x < WIDTH + 2 * SIZE_STRUCTURING_ELEMENT; x++) {
			host_input[y*WIDTH+x] = 0;
		}
	}

	/* Copiamos la imagen al buffer con el halo */
	for (y = 0; y < HEIGHT; y++) {
		for (x = 0; x < WIDTH; x++) {
			host_input[(y + SIZE_STRUCTURING_ELEMENT)*WIDTH+x + SIZE_STRUCTURING_ELEMENT] = input_image[y*WIDTH + x];
		}
	}

	start_time_inc_data = get_current_time();

	/*
	* Copiamos todos los arrays a la memoria de la GPU
	*/
	hipMemcpy( d_input, host_input, memSize, hipMemcpyHostToDevice);
	hipMemcpy( d_output, host_input, memSize, hipMemcpyHostToDevice);

	// Ejecutar dilatacion en la GPU
	/* Ejecuci�n kernel 2D */
	// TO DO - Calcular tama�o de bloque y grid para la correcta ejecucion del kernel
	// TO DO - Ejecutar el kernel
	
	// Copiamos de la memoria de la GPU a 
	hipMemcpy(gpu_output, d_input, memSize, hipMemcpyDeviceToHost);

	end_time_inc_data = get_current_time();

	checkCUDAError("Dilatacion CUDA: ");

	/*
	* Versi�n CPU dilatacion
	*/
	cpu_start_time = get_current_time();

	unsigned char temp;
	int numcols = WIDTH + SIZE_STRUCTURING_ELEMENT;
	
	for (y = 0; y < HEIGHT; y++) {
		for (x = 0; x < WIDTH; x++) {
			maxColor = 0;
			for (yWindow = -SIZE_STRUCTURING_ELEMENT; yWindow < SIZE_STRUCTURING_ELEMENT; yWindow++) {

				y2 = y + yWindow + SIZE_STRUCTURING_ELEMENT;
				
				for (xWindow = -SIZE_STRUCTURING_ELEMENT; xWindow < SIZE_STRUCTURING_ELEMENT; xWindow++) {
					x2 = x + xWindow + SIZE_STRUCTURING_ELEMENT;
				
					if (host_input[y2*WIDTH+x2]>maxColor)
						maxColor = host_input[y2*WIDTH + x2];
				}
			}
			host_output[(y + SIZE_STRUCTURING_ELEMENT)*WIDTH+x + SIZE_STRUCTURING_ELEMENT] = maxColor;
		}
	}
  
	cpu_end_time = get_current_time();

  /* Comprobamos que los resultados de la GPU coinciden con los calculados en la CPU */
  errors = 0;
  for (y = 0; y < HEIGHT; y++) {
    for (x = 0; x < WIDTH; x++) {
		if (host_input[(y + SIZE_STRUCTURING_ELEMENT)*WIDTH + x + SIZE_STRUCTURING_ELEMENT] != gpu_output[(y + SIZE_STRUCTURING_ELEMENT)*WIDTH + x + SIZE_STRUCTURING_ELEMENT]) {
        errors++;
        printf("Error en %d,%d (CPU=%i, GPU=%i)\n", x, y, \
			host_output[(y + SIZE_STRUCTURING_ELEMENT)*WIDTH+x + SIZE_STRUCTURING_ELEMENT], \
			gpu_output[(y + SIZE_STRUCTURING_ELEMENT)*WIDTH + x + SIZE_STRUCTURING_ELEMENT]);
      }
    }
  }
  
  if (errors == 0) printf("\n\n ***TEST CORRECTO*** \n\n\n");
  
  output_image = (unsigned char*)calloc(((WIDTH * HEIGHT) * 1), sizeof(unsigned char));

  for (y = 0; y < HEIGHT; y++) {
    for (x = 0; x < WIDTH; x++) {
		output_image[y*WIDTH + x] = host_output[(y + SIZE_STRUCTURING_ELEMENT)*WIDTH + x + SIZE_STRUCTURING_ELEMENT];
    }
  }

  hipFree(d_input);
  hipFree(d_output);

  printf("Tiempo ejecuci�n GPU (Incluyendo transferencia de datos): %fs\n", \
	 end_time_inc_data - start_time_inc_data);
  printf("Tiempo de ejecuci�n en la CPU                          : %fs\n", \
	 cpu_end_time - cpu_start_time);

	// Copiamos el resultado al formato de la libreria y guardamos el fichero BMP procesado
  for( int i=0 ; i < Image.TellHeight() ; i++)
  {
		for( int j=0 ; j < Image.TellWidth() ; j++)
		{
			Image(i,j)->Red = output_image[i*WIDTH+j];
			Image(i,j)->Green = output_image[i*WIDTH+j];
			Image(i,j)->Blue = output_image[i*WIDTH+j];
		}
  }
	// Guardamos el resultado de aplicar el filtro en un nuevo fichero
	Image.WriteToFile( "resultado_dilatacion.bmp" );

	getchar();
  return 0;
}


/* Funciones auxiliares */

#if _WIN32
	void getCurrentTimeStamp(timeStamp& _time)
	{
			QueryPerformanceCounter(&_time);
	}

	timeStamp getCurrentTimeStamp()
	{
			timeStamp tmp;
			QueryPerformanceCounter(&tmp);
			return tmp;
	}

	double getTimeMili()
	{
			timeStamp start;
			timeStamp dwFreq;
			QueryPerformanceFrequency(&dwFreq);
			QueryPerformanceCounter(&start);
			return double(start.QuadPart) / double(dwFreq.QuadPart);
	}
#endif 

double get_current_time()
{
	#if _WIN32 
		return getTimeMili();
	#else
		static int start = 0, startu = 0;
		struct timeval tval;
		double result;

		if (gettimeofday(&tval, NULL) == -1)
			result = -1.0;
		else if(!start) {
			start = tval.tv_sec;
			startu = tval.tv_usec;
			result = 0.0;
		}
		else
			result = (double) (tval.tv_sec - start) + 1.0e-6*(tval.tv_usec - startu);
		return result;
	#endif
}

/* Funci�n para comprobar errores CUDA */
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

